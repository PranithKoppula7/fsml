#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "hello.cuh"

__global__ void helloFromGPU(void) {
  printf("Hello World from GPU!\n");
}

namespace Hello {
  void hello_world() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceReset();
  }
}
