
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu(void) {
  printf("Hello World from GPU!\n");
}

int main(void) {
  printf("Hello World from CPU!\n");
  helloFromGpu<<<1, 10>>>();
  hipDeviceReset();
  return 0;
}
