#include "hip/hip_runtime.h"
#include "tensor.cuh"
#include <cstddef>
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_vector_types.h>

__global__ void tensorAdd(float* a, float* b, float* c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
  return;
}


namespace Tensor {
  float* tensor_add(float* a, float* b) {

    // calculate size
    int size = 0;
    for(int i = 0; i < sizeof(a); i++) {
      size++;
    }
    size_t nBytes = size * sizeof(float); 


    // host c 
    float* host_c = (float *)malloc(nBytes);

    // create pointers for gpu 
    float* cudaA = 0;
    float* cudaB = 0;
    float* cudaC = 0;

    // allocate memory in gpu 
    hipMalloc(&cudaA, nBytes);
    hipMalloc(&cudaB, nBytes);
    hipMalloc(&cudaC, nBytes);

    // copy vectors into gpu
    hipMemcpy(cudaA, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, nBytes, hipMemcpyHostToDevice);

    // gpu kernel method
    dim3 block (size);
    dim3 grid (size / block.x);
    tensorAdd<<<grid, block>>>(cudaA, cudaB, cudaC);
    printf("Excution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // copy from gpu -> cpu and return
    hipMemcpy(host_c, cudaC, nBytes, hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);

    return host_c; 
  }
}
