#include "hip/hip_runtime.h"
#include "ops.cuh"
#include <__clang_cuda_builtin_vars.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

__global__ void tensorAdd(float* a, float* b, float* c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
  return;
}

namespace ops {
  const float* tensor_add(float* a, float* b) {
    int c[sizeof(a) / sizeof(float)] = { 0 };
    
    // create pointers for gpu 
    float* cudaA = 0;
    float* cudaB = 0;
    float* cudaC = 0;

    // allocate memory in gpu 
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy vectors into gpu
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    // gpu kernel method
    tensorAdd<<<1, sizeof(a) / sizeof(float) >>>(cudaA, cudaB, cudaC);

    // copy from gpu -> cpu and return
    float* ans = 0;
    hipMemcpy(ans, cudaC, sizeof(cudaC), hipMemcpyDeviceToHost);

    return ans;
  }
}
