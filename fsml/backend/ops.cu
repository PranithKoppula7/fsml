#include "hip/hip_runtime.h"
#include "ops.cuh"
#include <cstdio>

__global__ void tensorAdd(const float* a, const float* b, float* c) {
  printf("Hello World from GPU!\n");
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
  return;
}


namespace Ops {
  float* tensor_add(float* a, float* b) {
    float c[sizeof(a) / sizeof(float)] = { 0 };

    // create pointers for gpu 
    float* cudaA = 0;
    float* cudaB = 0;
    float* cudaC = 0;

    // allocate memory in gpu 
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy vectors into gpu
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    // gpu kernel method
    tensorAdd<<<1, sizeof(a) / sizeof(float) >>>(cudaA, cudaB, cudaC);

    // copy from gpu -> cpu and return
    float* ans = 0;
    hipMemcpy(ans, cudaC, sizeof(cudaC), hipMemcpyDeviceToHost);

    hipDeviceReset();

    return ans;
  }
}
